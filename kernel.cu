
#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void VecAdd(int n, const float *A, const float *B, float* C) {

    /********************************************************************
     *
     * Compute C = A + B
     *   where A is a (1 * n) vector
     *   where B is a (1 * n) vector
     *   where C is a (1 * n) vector
     *
     ********************************************************************/
     
    /*************************************************************************/
    // INSERT CODE HERE
    int i = threadIdx.x + blockDim.x*blockIdx.x;				//Convert the 3 dimensional index to 1 dimension index for indexing 1D array.
    
    if(i<n)									// To limit the number of threads equal to 10,0000.
      C[i] = A[i] + B[i];							// Performing vector addition.
    return;
    /*************************************************************************/
	 
}

void basicVecAdd( float *A,  float *B, float *C, int n)
{

    // Initialize thread block and kernel grid dimensions 

    const unsigned int BLOCK_SIZE = 256; 

    /*************************************************************************/
    // INSERT CODE HERE
     dim3 dim_grid(((n-1)/BLOCK_SIZE)+1,1,1);					// Initializing the grid dimension using the ceiling function to get the number of blocks depending upon the total data.
     dim3 dim_block(BLOCK_SIZE*2,1,1);						// Initializing the number of threads in a block. As said in the lab1, 512 threads per block.
    VecAdd<<<dim_grid,dim_block>>>(n,A,B,C);					// Invoke the VecAdd function to be used by the GPU and perform vector addition.
    /*************************************************************************/
}

