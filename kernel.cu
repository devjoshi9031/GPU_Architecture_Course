
#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void VecAdd(int n, const float *A, const float *B, float* C) {

    /********************************************************************
     *
     * Compute C = A + B
     *   where A is a (1 * n) vector
     *   where B is a (1 * n) vector
     *   where C is a (1 * n) vector
     *
     ********************************************************************/
     
    /*************************************************************************/
    // INSERT CODE HERE


    /*************************************************************************/
	 
}

void basicVecAdd( float *A,  float *B, float *C, int n)
{

    // Initialize thread block and kernel grid dimensions ---------------------

    const unsigned int BLOCK_SIZE = 256; 

    /*************************************************************************/
    // INSERT CODE HERE


    /*************************************************************************/
}

